#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])



// ---- FP32 -----
// Warp Reduce Sum

// ————shfl_xor_sync  在warp内线程之间交换数据
// ————shfl_down_sync 在warp内同一线程交换数据
// xffffffff 线程掩码，表示warp中素有线程都参与
// mask 控制线程之间的数据交换  步长

// warp_reduce_sum_fp32 函数用于在一个warp内对浮点数进行归约求和操作
// 该函数使用了CUDA的线程间通信函数__shfl_xor_sync来实现线程之间的数据交换
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_fp32(float val){
    #pragma unroll // 指示编译器将循环展开（unroll）， 以提高性能
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}


// Block All Reduce Sum
// grid(N/256), block(256)
// a: Nx1, y = sum(a)
/*

- NUM_THREADS 是一个模版参数，用于定义 CUDA 内核中每个线程块的线程数量，是一个编译时常量，默认值为256 
- 通过使用模版参数 NUM_THREADS, 可以灵活地调整线程块的大小，而无需在代码中硬编码具体的线程数量。

在调用这个内核函数时，可以使用默认的 NUM_THREADS = 256, 那么每个线程块都会有 256 个线程。也可以调用时指定其他值：

block_all_reduce_sum_fp32_kernel<128><<<gridDim, blockDim>>>(a, y, N);

*/
template<const int NUM_THREADS = 256>
__global__ void block_all_reduce_sum_fp32_kernel(float* a, float* y, int N){
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE; // 计算每个block中的warp数量
    __shared__ float reduce_smem[NUM_WARPS]; // 每个block 共用一个 SM, 将结果保存到 SM 中
    
    // 将 数据加载到寄存器中
    float sum = (idx < N) ? a[idx] : 0.0f; // 读取数据

    int warp = tid / WARP_SIZE; // 计算当前线程所在的warp编号, 由于 threadIdx 是 一个 block 内的线程索引，所以可以通过除以 WARP_SIZE 来计算当前线程所在的 warp 编号
    int lane = tid % WARP_SIZE; // 计算当前线程在warp中的索引

    // 在warp内进行归约
    sum = warp_reduce_sum_fp32<WARP_SIZE>(sum); // 在warp内进行归约求和
    // 将每个warp的结果存储到共享内存中
    if (lane == 0){
        reduce_smem[warp] = sum; // 将每个warp的结果存储到共享内存中
    }
    __syncthreads(); // 同步所有线程，确保所有warp的结果都存储到共享内存中, 确保整个 block 进行规约时 可以实现
    // 在block内进行归约
    
    sum = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f; // 读取共享内存中的结果

    if(warp == 0) sum = warp_reduce_sum_fp32<WARP_SIZE>(sum); // 在第一个warp内进行归约求和
    if (tid == 0) atomicAdd(y, sum); // 将结果写入全局内存
}

// Block All Reduce Sum + float4 
// grid(N/256), block(256/4)
// a: Nx1, y = sum(a)
template<const int NUM_THREADS = 256>
__global__ void block_all_reduce_sum_f32x4_f32_kernel(float* a, float* y, int N){
    int tid = threadIdx.x;
    int idx = (blockIdx.x * NUM_THREADS + tid) * 4;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE; // 计算每个block中的warp数量


}
